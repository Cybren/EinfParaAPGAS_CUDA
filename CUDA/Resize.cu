#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "image.cuh"
#ifdef _WIN32
#include "hip/hip_runtime.h"
#endif // _WIN32

#define MIN(a,b) (((a)<(b))?(a):(b))
#define MAX(a,b) (((a)>(b))?(a):(b))
#define blockWidth1 32
#define blockHeight1 16

#define blockWidth2 128
#define blockHeight2 4

#define blocksize3 64 // 1080/15 = 72 => 64

struct container{
    int value;
    short xPos;
};

/*Notes
* types short instead of int? (width height, pixelEnergy) half of MemoryUsage
* #1: Order?
 */


int part(struct container list[], int left, int right) {
    unsigned int pivot = list[right].value;
    int x = (left - 1);
    for (int i = left; i < right; ++i) {
        if (list[i].value < pivot) {
            x++;
            struct container temp = list[i];
            list[i] = list[x];
            list[x] = temp;
        }
    }
    struct container temp = list[x + 1];
    list[x + 1] = list[right];
    list[right] = temp;
    return x + 1;
}


void quicksort(struct container* list, int left, int right) {
    if (left < right) {
        unsigned int pivot = part(list, left, right);

        quicksort(list, left, pivot - 1);
        quicksort(list, pivot + 1, right);
    }
}

__device__
struct container min(struct container container1, struct container container2) {
    return container1.value < container2.value ? container1 : container2;
}

__global__
void calcPixelEnergies(unsigned char* imageData, unsigned short* energyBuffer, int width, int height) {
    short bx = blockIdx.x;
    short by = blockIdx.y;
    short tx = threadIdx.x;
    short ty = threadIdx.y;
    short y = by * blockHeight1 + ty;
    short x = bx * blockWidth1 + tx;
    if (x == 0 && y == 0) {
        printf("calcPixelEnergies\n");
    }
    if (y < height && x < width) {
        unsigned int sum = 0;
        for (int i = -1; i < 2; i++) {
            for (int j = -1; j < 2; j++) {
                sum +=  abs(imageData[y * width * 3 + x * 3 + 0] - imageData[((y + j) % height) * width * 3 + ((x + i) % width) * 3 + 0]) + //R-Value
                        abs(imageData[y * width * 3 + x * 3 + 1] - imageData[((y + j) % height) * width * 3 + ((x + i) % width) * 3 + 1]) + //G-Value
                        abs(imageData[y * width * 3 + x * 3 + 2] - imageData[((y + j) % height) * width * 3 + ((x + i) % width) * 3 + 2]);  //B-Value
            }
        }
        energyBuffer[y * width + x] = sum;
    }
}

__global__
void calculateMinEnergySums(unsigned short* energyBuffer, struct container* sumBuffer, int width, int height) {
    short bx = blockIdx.x;
    short by = blockIdx.y;
    short tx = threadIdx.x;
    short ty = threadIdx.y;
    short y = by * blockHeight2 + ty;
    short x = bx * blockWidth2 + tx;
    struct container newContainer;
    if (x == 0 && y == 0) {
        printf("calculateMinEnergySums\n");
    }
    if (y < height && x < width) {
        if (x == width - 1) { // rightmost pixel of a row
            newContainer.value = energyBuffer[y * width + x] + MIN(energyBuffer[(y - 1) * width + x - 1], energyBuffer[(y - 1) * width + x]);
            newContainer.xPos = x;
        }else if (x == 0) { // leftmost pixel of a row
            newContainer.value = energyBuffer[y * width + x] + MIN(energyBuffer[(y - 1) * width + x], energyBuffer[(y - 1) * width + x + 1]);
            newContainer.xPos = x;
        }else {
            newContainer.value = energyBuffer[y * width + x] + MIN(MIN(energyBuffer[(y - 1) * width + x - 1], energyBuffer[(y - 1) * width + x]), energyBuffer[(y - 1) * width + x + 1]);
            newContainer.xPos = x;
        }
        sumBuffer[y * width + x] = newContainer;
    }
}

__global__
void increaseWidth(unsigned char* imageData, unsigned char* outputImageData, struct container* sumBuffer, unsigned short* seams, int numSeams, int inputWidth, int height) {
    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int threadNum = bx * blocksize3 + tx;
    int outputWidth = inputWidth + numSeams;
    if (threadNum == 0) {
        printf("increaseWidth numSeams: %d width: %d height: %d\n", numSeams, inputWidth, height);
    }
    //find seams
    if (threadNum < numSeams) {//sumBuffer sizeof(container) * input->width * input->height; seams sizeof(unsigned short) * input->height * numSeams
        seams[(height - 1) * numSeams + threadNum] = sumBuffer[inputWidth* (height-1) + threadNum].xPos;
        for (int y = height-2; y > -1; y--){
            int prevX = seams[threadNum * height + y + 1];
            if (prevX == inputWidth - 1) { // rightmost pixel of a row
                seams[y * numSeams + threadNum] = min(sumBuffer[y * inputWidth + prevX - 1], sumBuffer[y * inputWidth + prevX]).xPos;
            }else if (prevX == 0) { // leftmost pixel of a row
                seams[y * numSeams + threadNum] = min(sumBuffer[y * inputWidth + prevX], sumBuffer[y * inputWidth + prevX + 1]).xPos;
            }else {
                seams[y * numSeams + threadNum] = min(min(sumBuffer[y * inputWidth + prevX - 1], sumBuffer[y * inputWidth + prevX]), sumBuffer[y * inputWidth + prevX + 1]).xPos;
            }
        }
    }
    if (threadNum == 0) {
        printf("increaseWidth after seams\n");
    }
    __syncthreads();
    //create final Image
    if (threadNum < height) {//Mehr Threads m�glich (*3) und Jeder Thread eine Farbe
        int oldX = 0;
        int seamIndex = 0;
        int row = threadNum * outputWidth * 3;
        for (int x = 0; x < outputWidth; x++) {// illegal Memory Access
            /*if (oldX == seams[threadNum * width + seamIndex] && x > 0) {
                outputImageData[(threadNum * width) + x * 3]     = outputImageData[(threadNum * width) + (x - 1) * 3];
                outputImageData[(threadNum * width) + x * 3 + 1] = outputImageData[(threadNum * width) + (x - 1) * 3 + 1];
                outputImageData[(threadNum * width) + x * 3 + 2] = outputImageData[(threadNum * width) + (x - 1) * 3 + 2];
                seamIndex++;
            }else{*/
                outputImageData[row + x * 3]     = imageData[row + oldX * 3];
                outputImageData[row + x * 3 + 1] = imageData[row + oldX * 3 + 1];
                outputImageData[row + x * 3 + 2] = imageData[row + oldX * 3 + 2];
                oldX++;
            //}
        }
    }
    if (threadNum == 0) {
        printf("increaseWidth at end\n");
    }
}

int main(int argc, char* argv[]) {
    printf("start");
    if (argc != 4) {
        printf("Usage: %s inputJPEG outputJPEG numSeams\n", argv[0]);
        return 0;
    }
    char* inputFile = argv[1];
    char* outputFile = argv[2];
    int numSeams = atoi(argv[3]);
    //load image
    struct imgRawImage* input = loadJpegImageFile(inputFile);
    clock_t start = clock();
    //TO-DO use multiple GPUS
    //catch cuda errors
    hipError_t cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "hipSetDevice failed! ErrorCode %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus)); }

    int outputPixelBufferSize = (input->width + numSeams) * input->height;
    int inputPixelBufferSize = input->width * input->height;
    int sumBufferSize = sizeof(container) * input->width * input->height;
    unsigned char* d_inputImageData;
    unsigned char* d_outputImageData;
    unsigned short* d_energyBuffer;
    struct container* d_sumBuffer;
    struct container* sumBuffer;
    unsigned short* d_seams;

    //create outputimage struct
    struct imgRawImage* output;
    unsigned char* outputImageData = (unsigned char*)malloc(sizeof(unsigned char) * outputPixelBufferSize * 3);
    sumBuffer = (struct container*)malloc(sumBufferSize);
    output = (struct imgRawImage*)malloc(sizeof(struct imgRawImage));
    output->numComponents = input->numComponents;
    output->width = input->width + numSeams;
    output->height = input->height;

    //allocate nessessary memory on GPU
    cudaStatus = hipMalloc(&d_inputImageData, sizeof(unsigned char) * inputPixelBufferSize * 3);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "malloc d_inputImage failed! ErrorCode %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus)); }
    cudaStatus = hipMalloc(&d_outputImageData, sizeof(unsigned char) * outputPixelBufferSize * 3);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "malloc d_imageData failed! ErrorCode %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus)); }
    cudaStatus = hipMalloc(&d_energyBuffer, sizeof(unsigned short) * outputPixelBufferSize);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "malloc d_energyBuffer failed! ErrorCode %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus)); }
    cudaStatus = hipMalloc(&d_sumBuffer, sumBufferSize);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "malloc d_sumBuffer failed! ErrorCode %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus)); }
    cudaStatus = hipMalloc(&d_seams, sizeof(unsigned short) * input->height * numSeams);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "malloc d_seams failed! ErrorCode %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus)); }
    
    //start Kernel 1 to calculate all Energies
    cudaStatus = hipMemcpy(d_inputImageData, input->lpData, sizeof(unsigned char) * inputPixelBufferSize, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "Memory Copy input->lpData -> d_inputImageData failed! ErrorCode %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus)); }
    dim3 threadsPerBlock1(blockWidth1, blockHeight1);
    dim3 numBlocks1(ceil(input->width / threadsPerBlock1.x), ceil(input->height / threadsPerBlock1.y));
    calcPixelEnergies <<<numBlocks1, threadsPerBlock1>>>(d_inputImageData, d_energyBuffer, input->width, input->height);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) { fprintf(stderr, "calcPixelEnergies launch failed: %s\n", hipGetErrorString(cudaStatus)); }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize after launch calcPixelEnergies failed: %s\n", hipGetErrorString(cudaStatus)); }

    //start kernel2 to calculate the lowest energy-sums
    dim3 threadsPerBlock2(blockWidth2, blockHeight2);
    dim3 numBlocks2(ceil(input->width / threadsPerBlock2.x ), ceil(input->height / threadsPerBlock2.y));
    calculateMinEnergySums <<<numBlocks2, threadsPerBlock2>>>(d_energyBuffer, d_sumBuffer, input->width, input->height);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) { fprintf(stderr, "calculateMinEnergySums launch failed: %s\n", hipGetErrorString(cudaStatus)); }
    //hipFree(d_energyBuffer);
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize after launch calculateMinEnergySums failed: %s\n", hipGetErrorString(cudaStatus)); }

    //copy geht sehr viel kleiner, nur zu faul
    //calculate lowest energy-sums in last row on cpu
    printf("wanna calculate seams\n");
    /*cudaStatus = hipMemcpy(sumBuffer, d_sumBuffer, sumBufferSize, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "Memory Copy d_sumBuffer -> sumBuffer failed! ErrorCode %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus)); }
    quicksort(sumBuffer + (input->height - 1) * input->width * sizeof(container), 0, input->width);
    cudaStatus = hipMemcpy(d_sumBuffer, sumBuffer, sumBufferSize, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "Memory Copy sumBuffer -> d_sumBuffer failed! ErrorCode %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus)); }
    */
    //start final kernel to create outputImage
    dim3 threadsPerBlock3(blocksize3);
    dim3 numBlocks3(ceil(output->height / blocksize3));
    increaseWidth <<<numBlocks3, threadsPerBlock3 >>>(d_inputImageData, d_outputImageData, d_sumBuffer, d_seams, numSeams, input->width, input->height);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) { fprintf(stderr, "increaseWidth launch failed: %s\n", hipGetErrorString(cudaStatus)); }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize after launch increaseWidth failed: %s\n", hipGetErrorString(cudaStatus)); }
    printf("after Kernel3\n");
    //copy outputData to host
    cudaStatus = hipMemcpy(outputImageData, d_outputImageData, sizeof(unsigned char) * outputPixelBufferSize * 3, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "Memory Copy d_imageData -> outputImageData failed! ErrorCode %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus)); }
    output->lpData = outputImageData;
    //free all allocated Memory
    hipFree(&d_inputImageData);
    hipFree(&d_outputImageData);
    hipFree(&d_sumBuffer);
    hipFree(&d_seams);
    hipFree(&d_energyBuffer);
    clock_t end = clock();
    printf("Execution time: %4.2f sec\n", (double)((double)(end - start) / CLOCKS_PER_SEC));
    storeJpegImageFile(output, outputFile);
    return 0;
}

#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "image.cuh" 
#ifdef _WIN32
#include "hip/hip_runtime.h"
#endif // _WIN32

#define MIN(a,b) (((a)<(b))?(a):(b))
#define MAX(a,b) (((a)>(b))?(a):(b))

/*Notes
* types short instead of int? (width height, pixelEnergy) half of MemoryUsage
* #1: Order?
 */
/*__device__
unsigned int getColorDiff(unsigned char* imgData, int width, int x0, int y0, int x1, int y1) {
            //              Pixel0                                                  Pixel1
    return  abs(imgData[y0 * width * 3 + x0 * 3 + 0] - imgData[y1 * width * 3 + x1 * 3 + 0]) + //R-Value
            abs(imgData[y0 * width * 3 + x0 * 3 + 1] - imgData[y1 * width * 3 + x1 * 3 + 1]) + //G-Value
            abs(imgData[y0 * width * 3 + x0 * 3 + 2] - imgData[y1 * width * 3 + x1 * 3 + 2]);  //B-Value
}*/

/*__device__
unsigned int getPixelEnergy(unsigned char* imgData, int width, int height, int x, int y) {
    unsigned int sum = 0;
    for (int i = -1; i < 2; i++) {
        for (int j = -1; j < 2; j++) {
            //sum += getColorDiff(imgData, width, x, y, (x + i) % width , (y + j) % height);
            sum += abs(imgData[y * width * 3 + x * 3 + 0] - imgData[((y + j) % height) * width * 3 + ((x + i) % width) * 3 + 0]) + //R-Value
                abs(imgData[y * width * 3 + x * 3 + 1] - imgData[((y + j) % height) * width * 3 + ((x + i) % width) * 3 + 1]) + //G-Value
                abs(imgData[y * width * 3 + x * 3 + 2] - imgData[((y + j) % height) * width * 3 + ((x + i) % width) * 3 + 2]);  //B-Value
        }
    }
    return sum;
}*/

__device__
unsigned short* calcPixelEnergies(struct imgRawImage* image) {
    int width = image->width;
    int height = image->height;
    unsigned char* imgData = image->lpData;
    unsigned short* output = (unsigned short*)malloc(sizeof(unsigned short) * width * height);

    // #1
    for (int y = 0; y < height; y++){
        for (int x = 0; x < width; x++) {
            unsigned int sum = 0;
            for (int i = -1; i < 2; i++) {
                for (int j = -1; j < 2; j++) {
                    //sum += getColorDiff(imgData, width, x, y, (x + i) % width , (y + j) % height);
                    sum +=  abs(imgData[y * width * 3 + x * 3 + 0] - imgData[((y + j) % height) * width * 3 + ((x + i) % width) * 3 + 0]) + //R-Value
                            abs(imgData[y * width * 3 + x * 3 + 1] - imgData[((y + j) % height) * width * 3 + ((x + i) % width) * 3 + 1]) + //G-Value
                            abs(imgData[y * width * 3 + x * 3 + 2] - imgData[((y + j) % height) * width * 3 + ((x + i) % width) * 3 + 2]);  //B-Value
                }
            }
            //output[y * width + x] = getPixelEnergy(image->lpData, width, height, x, y);
            output[y * width + x] = sum;
        }
    }
    return output;
}

__device__
unsigned int* calculateMinEnergySums(unsigned short* pixelEnergies, int width, int height) {
    unsigned int* output = (unsigned int*)malloc(sizeof(unsigned int) * width * height);
    for (int y = 1; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            if (x == width - 1) { // rightmost pixel of a row
                output[y * width + x] = pixelEnergies[y * width + height] + MIN(output[(y - 1) * width + x - 1], output[(y - 1) * width + x]);
            }else if (x == 0) { // leftmost pixel of a row
                output[y * width + x] = pixelEnergies[y * width + height] + MIN(output[(y - 1) * width + x], output[(y - 1) * width + x + 1]);
            }else{
                output[y * width + x] = pixelEnergies[y * width + height] + MIN(MIN(output[(y - 1) * width + x - 1], output[(y - 1) * width + x]), output[(y - 1) * width + x + 1]);
            }
        }
    }
    return output;
}

__global__
void increaseWidth(struct imgRawImage* image, int numSeams) {
    int height = image->height;
    unsigned int* newMinEnergySums;
    unsigned char* newData;
    unsigned short* newPixelEnergies;

    unsigned short* pixelEnergies = calcPixelEnergies(image);
    unsigned int* minEnergySums = calculateMinEnergySums(pixelEnergies, image->width, image->height);

    // find seams by looking at the bottom row
    //unsigned int mins[numSeams]; doof
    unsigned int* mins = new unsigned int[numSeams];
    int width = image->width;
    for (int k = 0; k < numSeams; ++k) {
        mins[k] = 0;
        for (int j = 0; j < width; ++j) {
            int skip = 0;
            for (int l = 0; l < k; ++l) {
                if (mins[l] == j) {
                    skip = 1;
                    break;
                }
            }
            if (skip == 0 && minEnergySums[(height - 1) * width + j] < minEnergySums[(height - 1) * width + mins[k]]) {
                printf("m1=%i\n", minEnergySums[(height - 1)*width + j]);
                mins[k] = j;
            }
        }
    }

    for (int i = 0; i < numSeams; ++i) { 
        unsigned int minIdx = mins[i];
        // each iteration increases the width by 1
        int width = image->width;
        unsigned char* oldData = image->lpData;
        printf("iteration %i with width=%i and minIdx=%d\n", i, width, minIdx);
        newMinEnergySums = (unsigned int*) malloc(sizeof(unsigned int) * (width + 1) * height);
        newData = (unsigned char*) malloc(sizeof(unsigned char) * 3 * (width + 1) * height);
        newPixelEnergies = (unsigned short*) malloc(sizeof(unsigned int) * (width + 1) * height);

        // copy the pixels on the left side of the seam
        for (int j = 0; j <= minIdx; ++j) {
            newMinEnergySums[(height - 1) * (width + 1) + j] = minEnergySums[(height - 1) * width + j];
            newData[(height - 1) * (width + 1) * 3 + j * 3] = oldData[(height - 1) * width * 3 + j * 3];
            newData[(height - 1) * (width + 1) * 3 + j * 3 + 1] = oldData[(height - 1) * width * 3 + j * 3 + 1];
            newData[(height - 1) * (width + 1) * 3 + j * 3 + 2] = oldData[(height - 1) * width * 3 + j * 3 + 2];
            newPixelEnergies[(height - 1) * (width + 1) + j] = pixelEnergies[(height - 1) * width + j];
        }
        newMinEnergySums[(height - 1) * (width + 1) + minIdx + 1] = minEnergySums[(height - 1) * width + minIdx];
        newData[(height - 1) * (width + 1) * 3 + (minIdx + 1) * 3] = oldData[(height - 1) * width * 3 + minIdx * 3];
        newData[(height - 1) * (width + 1) * 3 + (minIdx + 1) * 3 + 1] = oldData[(height - 1) * width * 3 + minIdx * 3 + 1];
        newData[(height - 1) * (width + 1) * 3 + (minIdx + 1) * 3 + 2] = oldData[(height - 1) * width * 3 + minIdx * 3 + 2];
        newPixelEnergies[(height - 1) * (width + 1) + minIdx + 1] = pixelEnergies[(height - 1) * width + minIdx];
        // move all pixels right of the seam 1 to the right
        for (int j = minIdx + 1; j < width; ++j) {
            newMinEnergySums[(height - 1) * (width + 1) + j + 1] = minEnergySums[(height - 1) * width + j];
            newData[(height - 1) * (width + 1) * 3 + (j + 1) * 3] = oldData[(height - 1) * width * 3 + j * 3];
            newData[(height - 1) * (width + 1) * 3 + (j + 1) * 3 + 1] = oldData[(height - 1) * width * 3 + j * 3 + 1];
            newData[(height - 1) * (width + 1) * 3 + (j + 1) * 3 + 2] = oldData[(height - 1) * width * 3 + j * 3 + 2];
            newPixelEnergies[(height - 1) * (width + 1) + j + 1] = pixelEnergies[(height - 1) * width + j];
        }
        int x = minIdx;
        for (int y = height - 2; y >= 0; --y) {
            unsigned int min;
            if (x == 0) {
                min = MIN(minEnergySums[y * width + x], minEnergySums[y * width + x + 1]);
            }else if (x == width - 1) {
                min = MIN(minEnergySums[y * width + x - 1], minEnergySums[y * width + x]);
            }else {
                min = MIN(minEnergySums[y * width + x - 1], MIN(minEnergySums[y * width + x], minEnergySums[y * width + x + 1]));
            }

            if (x > 0 && minEnergySums[y*width + (x - 1)] == min) {
                x = x - 1;
            }else if (x <= width - 1 && minEnergySums[y * width + (x + 1)] == min) {
                x = x + 1;
            }

            for (int j = 0; j <= x; ++j) {
                newMinEnergySums[y * (width + 1) + j] = minEnergySums[y * width + j];
                newData[y * (width + 1) * 3 + j * 3] = oldData[y * width * 3 + j * 3];
                newData[y * (width + 1) * 3 + j * 3 + 1] = oldData[y * width * 3 + j * 3 + 1];
                newData[y * (width + 1) * 3 + j * 3 + 2] = oldData[y * width * 3 + j * 3 + 2];
                newPixelEnergies[y * (width + 1) + j] = pixelEnergies[y * width + j];
            }
            newMinEnergySums[y * (width + 1) + x + 1] = minEnergySums[y * width + x];
            newData[y * (width + 1) * 3 + (x + 1) * 3] = oldData[y * width * 3 + x * 3];
            newData[y * (width + 1) * 3 + (x + 1) * 3 + 1] = oldData[y * width * 3 + x * 3 + 1];
            newData[y * (width + 1) * 3 + (x + 1) * 3 + 2] = oldData[y * width * 3 + x * 3 + 2];
            newPixelEnergies[y * (width + 1) + x + 1] = pixelEnergies[y * width + x];
            for (int j = x + 1; j < width; ++j) {
                newMinEnergySums[y * (width + 1) + j + 1] = minEnergySums[y * width + j];
                newData[y * (width + 1) * 3 + (j + 1) * 3] = oldData[y * width * 3 + j * 3];
                newData[y * (width + 1) * 3 + (j + 1) * 3 + 1] = oldData[y * width * 3 + j * 3 + 1];
                newData[y * (width + 1) * 3 + (j + 1) * 3 + 2] = oldData[y * width * 3 + j * 3 + 2];
                newPixelEnergies[y * (width + 1) + j + 1] = pixelEnergies[y * width + j];
            }
        }
        free(image->lpData);
        image->lpData = newData;
        image->width = width + 1;
        free(minEnergySums);
        free(pixelEnergies);
        pixelEnergies = newPixelEnergies;
        minEnergySums = newMinEnergySums;
    }
}

int main(int argc, char* argv[]) {
    if (argc != 4) {
        printf("Usage: %s inputJPEG outputJPEG numSeams\n", argv[0]);
        return 0;
    }
    char* inputFile = argv[1];
    char* outputFile = argv[2];
    int numSeams = atoi(argv[3]);
    struct imgRawImage* d_inputImage, d_outputImage;

    struct imgRawImage* input = loadJpegImageFile(inputFile);

    struct imgRawImage* output;
    int pixelBufferSize = (input->width + numSeams )* input->height * 3;
    unsigned char* imageData = (unsigned char*)malloc(sizeof(unsigned char) * pixelBufferSize);
    output = (struct imgRawImage*)malloc(sizeof(struct imgRawImage));
    output->numComponents = input->numComponents;
    output->width = input->width + numSeams;
    output->height = input->height;
    output->lpData = imageData;

    clock_t start = clock();
    hipMalloc(&d_inputImage, sizeof(input));
    hipMalloc(&d_outputImage, sizeof(output));

    dim3 threadsPerBlock = (128, 4); //512 Threads per Block (max 3 Blocks per SM)
    dim3 numBlocks = ((output->width + threadsPerBlock.x - 1) / threadsPerBlock.x, (output->height + threadsPerBlock.y - 1) / threadsPerBlock.y); // at least 15*270 = 4050 blocks
    increaseWidth<<<numBlocks, threadsPerBlock>>>(d_inputImage, numSeams);
    struct imgRawImage* outputImage = hipMemcpy(output, d_outputImage, sizeof(d_outputImage), hipMemcpyDeviceToHost);
    hipFree(&d_inputImage);
    hipFree(&d_outputImage);
    clock_t end = clock();
    printf("Execution time: %4.2f sec\n", (double)((double)(end - start) / CLOCKS_PER_SEC));
    storeJpegImageFile(output, outputFile);
    return 0;
}

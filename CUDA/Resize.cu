#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "image.cuh" 

#define MIN(a,b) (((a)<(b))?(a):(b))
#define MAX(a,b) (((a)>(b))?(a):(b))

/*Notes
* image global?
* types short instead of int? (width height, pixelEnergy) half of MemoryUsage
* #1: Order?
* #2: Only consider the numSeems lowest PixelEnergies for each step
 */
unsigned int getColorDiff(unsigned char* imgData, int width, int x0, int y0, int x1, int y1) {
            //              Pixel0                                                  //Pixel1
    return  abs(imgData[y0 * width * 3 + x0 * 3 + 0] - imgData[y1 * width * 3 + x1 * 3 + 0]) + //R-Value
            abs(imgData[y0 * width * 3 + x0 * 3 + 1] - imgData[y1 * width * 3 + x1 * 3 + 1]) + //G-Value
            abs(imgData[y0 * width * 3 + x0 * 3 + 2] - imgData[y1 * width * 3 + x1 * 3 + 2]);  //B-Value
}

unsigned int getPixelEnergy(unsigned char* imgData, int width, int height, int x, int y) {
    unsigned int sum = 0;
    for (int i = -1; i < 2; i++){
        for (int j = -1; j < 2; j++) {
            sum += getColorDiff(imgData, width, x, y, (x + i) % width , (y + j) % height);
        }
    }
    return sum;
}

unsigned int* calcPixelEnergies(struct imgRawImage* image) {
    int width = image->width;
    int height = image->height;
    unsigned int* output = (unsigned int*)malloc(sizeof(unsigned int) * width * height);
    // #1
    for (int y = 0; y < height; y++){
        for (int x = 0; x < width; x++) {
            output[y * width + x] = getPixelEnergy(image->lpData, width, height, x, y);
        }
    }
    return output;
}

// #2
unsigned int* calculateMinEnergySums(unsigned int* pixelEnergies, int width, int height) {
    unsigned int* output = (unsigned int*)malloc(sizeof(unsigned int) * width * height);
    for (int y = 1; y < height; ++y) {
        for (int x = 0; x < width; ++x) {
            if (x == width - 1) { // rightmost pixel of a row
                output[y * width + x] = pixelEnergies[y * width + height] + MIN(output[(y - 1) * width + x - 1], output[(y - 1) * width + x]);
            }else if (x == 0) { // leftmost pixel of a row
                output[y * width + x] = pixelEnergies[y * width + height] + MIN(output[(y - 1) * width + x], output[(y - 1) * width + x + 1]);
            }else{
                output[y * width + x] = pixelEnergies[y * width + height] + MIN(MIN(output[(y - 1) * width + x - 1], output[(y - 1) * width + x]), output[(y - 1) * width + x + 1]);
            }
        }
    }
    return output;
}

struct imgRawImage* increaseWidth(struct imgRawImage* image, int numSeams) {
    int height = image->height;
    unsigned int* newMinEnergySums;
    unsigned char* newData;
    unsigned int* newPixelEnergies;

    unsigned int* pixelEnergies = calcPixelEnergies(image);
    unsigned int* minEnergySums = calculateMinEnergySums(pixelEnergies, image->width, image->height);

    // find seams by looking at the bottom row
    //unsigned int mins[numSeams]; doof
    unsigned int* mins = new unsigned int[numSeams];
    int width = image->width;
    for (int k = 0; k < numSeams; ++k) {
        mins[k] = 0;
        for (int j = 0; j < width; ++j) {
            int skip = 0;
            for (int l = 0; l < k; ++l) {
                if (mins[l] == j) {
                    skip = 1;
                    break;
                }
            }
            if (skip == 0 && minEnergySums[(height - 1) * width + j] < minEnergySums[(height - 1) * width + mins[k]]) {
                //printf("m1=%i\n", m1(height - 1, j));
                printf("m1=%i\n", minEnergySums[(height - 1)*width + j]);
                mins[k] = j;
            }
        }
    }

    for (int i = 0; i < numSeams; ++i) {
        unsigned int minIdx = mins[i];
        // each iteration increases the width by 1
        int width = image->width;
        unsigned char* oldData = image->lpData;
        printf("iteration %i with width=%i and minIdx=%d\n", i, width, minIdx);
        newMinEnergySums = (unsigned int*) malloc(sizeof(unsigned int) * (width + 1) * height);
        newData = (unsigned char*) malloc(sizeof(unsigned char) * 3 * (width + 1) * height);
        newPixelEnergies = (unsigned int*) malloc(sizeof(unsigned int) * (width + 1) * height);

        // copy the pixels on the left side of the seam
        for (int j = 0; j <= minIdx; ++j) {
            //nw(height - 1, j) = m1(height - 1, j);
            newMinEnergySums[(height - 1) * (width + 1) + j] = minEnergySums[(height - 1) * width + j];
            //nd3(height - 1, j, 0) = od3(height - 1, j, 0);
            newData[(height - 1) * (width + 1) * 3 + j * 3] = oldData[(height - 1) * width * 3 + j * 3];
            //nd3(height - 1, j, 1) = od3(height - 1, j, 1);
            newData[(height - 1) * (width + 1) * 3 + j * 3 + 1] = oldData[(height - 1) * width * 3 + j * 3 + 1];
            //nd3(height - 1, j, 2) = od3(height - 1, j, 2);
            newData[(height - 1) * (width + 1) * 3 + j * 3 + 2] = oldData[(height - 1) * width * 3 + j * 3 + 2];
            //ng(height - 1, j) = g(height - 1, j);
            newPixelEnergies[(height - 1) * (width + 1) + j] = pixelEnergies[(height - 1) * width + j];
        }
        //nw(height - 1, minIdx + 1) = m1(height - 1, minIdx);
        newMinEnergySums[(height - 1) * (width + 1) + minIdx + 1] = minEnergySums[(height - 1) * width + minIdx];
        //nd3(height - 1, minIdx + 1, 0) = od3(height - 1, minIdx, 0);
        newData[(height - 1) * (width + 1) * 3 + (minIdx + 1) * 3] = oldData[(height - 1) * width * 3 + minIdx * 3];
        //nd3(height - 1, minIdx + 1, 1) = od3(height - 1, minIdx, 1);
        newData[(height - 1) * (width + 1) * 3 + (minIdx + 1) * 3 + 1] = oldData[(height - 1) * width * 3 + minIdx * 3 + 1];
        //nd3(height - 1, minIdx + 1, 2) = od3(height - 1, minIdx, 2);
        newData[(height - 1) * (width + 1) * 3 + (minIdx + 1) * 3 + 2] = oldData[(height - 1) * width * 3 + minIdx * 3 + 2];
        //ng(height - 1, minIdx + 1) = g(height - 1, minIdx);
        newPixelEnergies[(height - 1) * (width + 1) + minIdx + 1] = pixelEnergies[(height - 1) * width + minIdx];
        // move all pixels right of the seam 1 to the right
        for (int j = minIdx + 1; j < width; ++j) {
            //nw(height - 1, j + 1) = m1(height - 1, j);
            newMinEnergySums[(height - 1) * (width + 1) + j + 1] = minEnergySums[(height - 1) * width + j];
            //nd3(height - 1, j + 1, 0) = od3(height - 1, j, 0);
            newData[(height - 1) * (width + 1) * 3 + (j + 1) * 3] = oldData[(height - 1) * width * 3 + j * 3];
            //nd3(height - 1, j + 1, 1) = od3(height - 1, j, 1);
            newData[(height - 1) * (width + 1) * 3 + (j + 1) * 3 + 1] = oldData[(height - 1) * width * 3 + j * 3 + 1];
            //nd3(height - 1, j + 1, 2) = od3(height - 1, j, 2);
            newData[(height - 1) * (width + 1) * 3 + (j + 1) * 3 + 2] = oldData[(height - 1) * width * 3 + j * 3 + 2];
            //ng(height - 1, j + 1) = g(height - 1, j);
            newPixelEnergies[(height - 1) * (width + 1) + j + 1] = pixelEnergies[(height - 1) * width + j];
        }
        int x = minIdx;
        for (int y = height - 2; y >= 0; --y) {
            unsigned int min;
            if (x == 0) {
                //min = MIN(m1(y, x), m1(y, x + 1));
                min = MIN(minEnergySums[y * width + x], minEnergySums[y * width + x + 1]);
            }else if (x == width - 1) {
                //min = MIN(m1(y, x - 1), m1(y, x));
                min = MIN(minEnergySums[y * width + x - 1], minEnergySums[y * width + x]);
            }else {
                //min = MIN(m1(y, x - 1), MIN(m1(y, x), m1(y, x + 1)));
                min = MIN(minEnergySums[y * width + x - 1], MIN(minEnergySums[y * width + x], minEnergySums[y * width + x + 1]));
            }

            //if (x > 0 && m1(y, x - 1) == min) {
            if (x > 0 && minEnergySums[y*width + (x - 1)] == min) {
                x = x - 1;
            //}else if (x <= width - 1 && m1(y, x + 1) == min) {
            }else if (x <= width - 1 && minEnergySums[y * width + (x + 1)] == min) {
                x = x + 1;
            }
            for (int j = 0; j <= x; ++j) {
                //nw(y, j) = m1(y, j);
                newMinEnergySums[y * (width + 1) + j] = minEnergySums[y * width + j];
                //nd3(y, j, 0) = od3(y, j, 0);
                newData[y * (width + 1) * 3 + j * 3] = oldData[y * width * 3 + j * 3];
                //nd3(y, j, 1) = od3(y, j, 1);
                newData[y * (width + 1) * 3 + j * 3 + 1] = oldData[y * width * 3 + j * 3 + 1];
                //nd3(y, j, 2) = od3(y, j, 2);
                newData[y * (width + 1) * 3 + j * 3 + 2] = oldData[y * width * 3 + j * 3 + 2];
                //ng(y, j) = g(y, j);
                newPixelEnergies[y * (width + 1) + j] = pixelEnergies[y * width + j];
            }
            //nw(y, x + 1) = m1(y, x);
            newMinEnergySums[y * (width + 1) + x + 1] = minEnergySums[y * width + x];
            //nd3(y, x + 1, 0) = od3(y, x, 0);
            newData[y * (width + 1) * 3 + (x + 1) * 3] = oldData[y * width * 3 + x * 3];
            //nd3(y, x + 1, 1) = od3(y, x, 1);
            newData[y * (width + 1) * 3 + (x + 1) * 3 + 1] = oldData[y * width * 3 + x * 3 + 1];
            //nd3(y, x + 1, 2) = od3(y, x, 2);
            newData[y * (width + 1) * 3 + (x + 1) * 3 + 2] = oldData[y * width * 3 + x * 3 + 2];
            //ng(y, x + 1) = g(y, x);
            newPixelEnergies[y * (width + 1) + x + 1] = pixelEnergies[y * width + x];
            for (int j = x + 1; j < width; ++j) {
                //nw(y, j + 1) = m1(y, j);
                newMinEnergySums[y * (width + 1) + j + 1] = minEnergySums[y * width + j];
                //nd3(y, j + 1, 0) = od3(y, j, 0);
                newData[y * (width + 1) * 3 + (j + 1) * 3] = oldData[y * width * 3 + j * 3];
                //nd3(y, j + 1, 1) = od3(y, j, 1);
                newData[y * (width + 1) * 3 + (j + 1) * 3 + 1] = oldData[y * width * 3 + j * 3 + 1];
                //nd3(y, j + 1, 2) = od3(y, j, 2);
                newData[y * (width + 1) * 3 + (j + 1) * 3 + 2] = oldData[y * width * 3 + j * 3 + 2];
                //ng(y, j + 1) = g(y, j);
                newPixelEnergies[y * (width + 1) + j + 1] = pixelEnergies[y * width + j];
            }
        }
        free(image->lpData);
        image->lpData = newData;
        image->width = width + 1;
        free(minEnergySums);
        free(pixelEnergies);
        pixelEnergies = newPixelEnergies;
        minEnergySums = newMinEnergySums;
    }
    return image;
}

int main(int argc, char* argv[]) {
    if (argc != 4) {
        printf("Usage: %s inputJPEG outputJPEG numSeams\n", argv[0]);
        return 0;
    }
    char* inputFile = argv[1];
    char* outputFile = argv[2];
    int numSeams = atoi(argv[3]);

    struct imgRawImage* input = loadJpegImageFile(inputFile);
    clock_t start = clock();

    struct imgRawImage* output = increaseWidth(input, numSeams);

    clock_t end = clock();
    printf("Execution time: %4.2f sec\n", (double)((double)(end - start) / CLOCKS_PER_SEC));
    storeJpegImageFile(output, outputFile);
    return 0;
}

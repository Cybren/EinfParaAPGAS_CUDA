#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <stdbool.h>
#include "image.cuh"
#ifdef _WIN32
#include "hip/hip_runtime.h"
#endif // _WIN32

#define blockWidth1 32
#define blockHeight1 16

#define blockWidth2 128

#define blocksize3 32

#define blocksize4 64 // 1080/15 = 72 => 64

struct container {
    unsigned int value;
    int xPos;
};

//quicksort for values
int partValue(struct container list[], int left, int right) {
    int pivot = list[right].value;
    int x = (left - 1);
    for (int i = left; i < right; ++i) {
        if (list[i].value <= pivot) {
            x++;
            struct container temp = list[i];
            list[i] = list[x];
            list[x] = temp;
        }
    }
    struct container temp = list[x + 1];
    list[x + 1] = list[right];
    list[right] = temp;
    return x + 1;
}


void quicksortValue(struct container* list, int left, int right) {
    if (left < right) {
        unsigned int pivot = partValue(list, left, right);

        quicksortValue(list, left, pivot - 1);
        quicksortValue(list, pivot + 1, right);
    }
}

//wuicksort for int
int partint(unsigned int list[], int left, int right) {
    unsigned int pivot = list[right];
    int x = (left - 1);
    for (int i = left; i < right; ++i) {
        if (list[i] < pivot) {
            x++;
            unsigned int temp = list[i];
            list[i] = list[x];
            list[x] = temp;
        }
    }
    unsigned int temp = list[x + 1];
    list[x + 1] = list[right];
    list[right] = temp;
    return x + 1;
}


void quicksortint(unsigned int* list, int left, int right) {
    if (left < right) {
        int pivot = partint(list, left, right);

        quicksortint(list, left, pivot - 1);
        quicksortint(list, pivot + 1, right);
    }
}

//helpermethods
__device__
unsigned int MIN(int a, int b) {
    return a < b ? a : b;
}

__device__
int MAX(int a, int b) {
    return a > b ? a : b;
}

__device__
int MOD(int a, int b) {
    return ((a % b )+b) % b;
}

__device__
struct container minContainer(struct container container1, struct container container2) {
    return container1.value < container2.value ? container1 : container2;
}

__global__
void debug(unsigned int* seams, int height, int numSeams) {
    for (int y = 0; y < height; y++){
        for (int x = 0; x < numSeams; x++){
            printf("(%d;%d): %d", x, y, seams[y * numSeams + x]);
        }
        printf("\n");
    }
}

//checked
__global__
void calculatePixelEnergies(unsigned char* inputData, unsigned int* pixelEnergies, int width, int height) {
    //__shared__ unsigned int inputTile[blockHeight1+2][blockWidth1+2];
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int y = by * blockHeight1 + ty;
    int x = bx * blockWidth1 + tx;
    int sum;
    int actualY;
    int actualX;
    if (y < height && x < width) {
        sum = 0;
        for (int offsetX = -1; offsetX < 2; offsetX++) {
            for (int offsetY = -1; offsetY < 2; offsetY++) {
                actualY = MOD((y + offsetY), height);
                actualX = MOD((x + offsetX), width);
                sum += abs(inputData[(y * width + x) * 3] - inputData[(actualY * width + actualX) * 3])
                    + abs(inputData[(y * width + x) * 3 + 1] - inputData[(actualY * width + actualX) * 3 + 1])
                    + abs(inputData[(y * width + x) * 3 + 2] - inputData[(actualY * width + actualX) * 3 + 2]);
            }
        }
        pixelEnergies[y * width + x] = sum;
    }
}

//checked
__global__
void calculateMinEnergySums(unsigned int* pixelEnergies, struct container* minEnergySums, int width, int row) {
    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int x = bx * blockWidth2 + tx;
    //use "tiling"
    __shared__ unsigned int tiledMinEnergySums[blockWidth2];
    if (x < width) {
        if (row == 0) {
            struct container newContainer;
            newContainer.value = pixelEnergies[x];
            newContainer.xPos = x;
            minEnergySums[x] = newContainer;
        }else {
            tiledMinEnergySums[tx] = minEnergySums[(row - 1) * width + x].value;
            __syncthreads();
            struct container newContainer;
            if (x == 0) { // leftmost pixel of a row
                newContainer.value = pixelEnergies[row * width + x] + MIN(tiledMinEnergySums[tx], (tx + 1 < blockWidth2) ? tiledMinEnergySums[tx + 1] : minEnergySums[(row - 1) * width + x + 1].value);
            }else if (x == width - 1) { // rightmost pixel of a row 
                newContainer.value = pixelEnergies[row * width + x] + MIN((tx - 1 > 0) ? tiledMinEnergySums[tx - 1] : minEnergySums[(row - 1) * width + x - 1].value, tiledMinEnergySums[tx]);
            }else {
                newContainer.value = pixelEnergies[row * width + x] + MIN(MIN((tx - 1 > 0) ? tiledMinEnergySums[tx - 1] : minEnergySums[(row - 1) * width + x - 1].value, tiledMinEnergySums[tx]), (tx + 1 < blockWidth2) ? tiledMinEnergySums[tx + 1] : minEnergySums[(row - 1) * width + x + 1].value);
            }
            newContainer.xPos = x;
            minEnergySums[row * width + x] = newContainer;
        }
    }
}
__global__
void calcSeams(struct container* minEnergySums, unsigned int* seams, int inputWidth, int height, int numSeams) {
    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int threadNum = bx * blocksize3 + tx;
    if (threadNum < numSeams) {
        for (int y = height - 2; y > -1; y--) {
            unsigned int prevX = seams[(y + 1) * numSeams + threadNum];
            if (prevX == inputWidth - 1) { // rightmost pixel of a row
                seams[y * numSeams + threadNum] = minContainer(minEnergySums[y * inputWidth + prevX - 1], minEnergySums[y * inputWidth + prevX]).xPos;
            }else if (prevX == 0) { // leftmost pixel of a row
                seams[y * numSeams + threadNum] = minContainer(minEnergySums[y * inputWidth + prevX], minEnergySums[y * inputWidth + prevX + 1]).xPos;
            }else {
                seams[y * numSeams + threadNum] = minContainer(minContainer(minEnergySums[y * inputWidth + prevX - 1], minEnergySums[y * inputWidth + prevX]), minEnergySums[y * inputWidth + prevX + 1]).xPos;
            }
        }
    }
}

__global__
void increaseWidth(unsigned char *inputData, unsigned char *outputData, struct container* minEnergySums, unsigned int* seams, int inputWidth, int height, int numSeams) {
    int bx = blockIdx.x;
    int tx = threadIdx.x;
    int threadNum = bx * blocksize4 + tx;
    if(threadNum < height){
        //int to track where we are in the old picture
        int oldX = -1;
        int seamIndex = 0;
        int outputRow = threadNum * (inputWidth + numSeams) * 3;
        int inputRow = threadNum * inputWidth * 3;
        for (int x = 0; x < (inputWidth + numSeams); x++) {
            /*bool condition = (x > 0 && oldX == seams[threadNum * numSeams + seamIndex] && seamIndex < numSeams);
            oldX = condition ? oldX: oldX + 1;
            outputData[row + x * 3] = condition ? outputData[row + (x - 1) * 3] : inputData[inputRow + oldX * 3];
            outputData[row + x * 3 + 1] = condition ? outputData[row + (x - 1) * 3 + 1] : inputData[inputRow + oldX * 3 + 1];
            outputData[row + x * 3 + 2] = condition ? outputData[row + (x - 1) * 3 + 2] : inputData[inputRow + oldX * 3 + 2];
            seamIndex = condition ? seamIndex + 1 : seamIndex;*/
            //copy last pixel if oldX is at a point where a seam is 
            if (x > 0 && oldX == seams[threadNum * numSeams + seamIndex] && seamIndex < numSeams) {
                outputData[outputRow + x * 3] = outputData[outputRow + (x - 1) * 3];
                outputData[outputRow + x * 3 + 1] = outputData[outputRow + (x - 1) * 3 + 1];
                outputData[outputRow + x * 3 + 2] = outputData[outputRow + (x - 1) * 3 + 2];
                seamIndex++;
            }else {
            //else just copy the pixel of the old picture
                oldX++;
                outputData[outputRow + x * 3] = inputData[inputRow + oldX * 3];
                outputData[outputRow + x * 3 + 1] = inputData[inputRow + oldX * 3 + 1];
                outputData[outputRow + x * 3 + 2] = inputData[inputRow + oldX * 3 + 2];
            }
        }
    }
}

int main(int argc, char* argv[]) {
    if (argc < 4) {
        printf("Usage: %s inputJPEG outputJPEG numSeams\n", argv[0]);
        return 0;
    }
    char* inputFile = argv[1];
    char* outputFile = argv[2];
    int numSeams = atoi(argv[3]);

    struct imgRawImage* input = loadJpegImageFile(inputFile);
    clock_t start = clock();
    //host
    int width = input->width;
    int height = input->height;


    size_t inputDataSize_t = sizeof(unsigned char) * width * height * 3;
    size_t outputDataSize_t = sizeof(unsigned char) * (width + numSeams)* height * 3;
    size_t pixelEnergiesSize_t = sizeof(unsigned int) * width * height;
    size_t minEnergySumsSize_t = sizeof(struct container) * height * width;
    size_t seamsSize_t = sizeof(unsigned int) * numSeams * height;
    size_t seamStartSize_t = sizeof(unsigned int) * numSeams;
    size_t lastMinEnergySumsSize_t = sizeof(struct container) * width;

    unsigned int* seamsStart = (unsigned int*)malloc(seamStartSize_t);
    struct container* lastMinEnergySums = (struct container*)malloc(lastMinEnergySumsSize_t);
    unsigned char* outputData = (unsigned char*)malloc(outputDataSize_t);
    hipError_t cudaStatus;
    //device
    unsigned char* d_inputData;
    unsigned char* d_outputData;
    unsigned int* d_pixelEnergies;
    struct container* d_minEnergySums;
    unsigned int* d_seams;

    //allocate Devicememory
    cudaStatus = hipMalloc(&d_inputData, inputDataSize_t);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "malloc d_inputData failed! ErrorCode %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus)); }
    hipMemset(d_inputData, 0, inputDataSize_t);
    cudaStatus = hipMalloc(&d_outputData, outputDataSize_t);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "malloc d_inputImage failed! ErrorCode %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus)); }
    hipMemset(d_outputData, 0, outputDataSize_t);
    cudaStatus = hipMalloc(&d_pixelEnergies, pixelEnergiesSize_t);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "malloc d_pixelEnergies failed! ErrorCode %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus)); }
    hipMemset(d_pixelEnergies, 0, pixelEnergiesSize_t);
    cudaStatus = hipMalloc(&d_minEnergySums, minEnergySumsSize_t);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "malloc d_minEnergySums failed! ErrorCode %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus)); }
    hipMemset(d_minEnergySums, 0, minEnergySumsSize_t);
    cudaStatus = hipMalloc(&d_seams, seamsSize_t);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "malloc d_minEnergySums failed! ErrorCode %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus)); }
    hipMemset(d_seams, 0, seamsSize_t);

    //start kernel1 calculatePixelEnergies
    cudaStatus = hipMemcpy(d_inputData, input->lpData, inputDataSize_t, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "Memory Copy input->lpData -> d_inputData failed! ErrorCode %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus)); }
    dim3 threadsPerBlock1(blockWidth1, blockHeight1);
    dim3 numBlocks1(ceil(width / (double)threadsPerBlock1.x), ceil(height / (double)threadsPerBlock1.y));

    calculatePixelEnergies<<<numBlocks1, threadsPerBlock1>>>(d_inputData, d_pixelEnergies, width, height);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) { fprintf(stderr, "calculatePixelEnergies launch failed: %s\n", hipGetErrorString(cudaStatus)); }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize after launch calculatePixelEnergies failed: %s\n", hipGetErrorString(cudaStatus)); }

    //start kernel2 calculateMinEnergySums
    dim3 threadsPerBlock2(blockWidth2);
    dim3 numBlocks2(ceil(width / (double)threadsPerBlock2.x));

    for (int i = 0; i < height; i++){
        calculateMinEnergySums << <numBlocks2, threadsPerBlock2 >> > (d_pixelEnergies, d_minEnergySums, width, i);
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize after launch calculateMinEnergySums failed: %s\n", hipGetErrorString(cudaStatus)); }
    }
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) { fprintf(stderr, "calculateMinEnergySums launch failed: %s\n", hipGetErrorString(cudaStatus)); }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize after launch calculateMinEnergySums failed: %s\n", hipGetErrorString(cudaStatus)); }

    //calculate Seams schauen wegen k>width
    cudaStatus = hipMemcpy(lastMinEnergySums, d_minEnergySums + width * (height - 1), lastMinEnergySumsSize_t, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "Memory Copy d_minEnergySums -> lastMinEnergySums failed! ErrorCode %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus)); }

    //sort by value
    quicksortValue(lastMinEnergySums, 0, width - 1);
    /*for (int i = 0; i < numSeams; i++) {
        seamsStart[i] = lastMinEnergySums[i].xPos;
    }*/
    int seamIndex = 0;
    int minSumIndex = 0;
    while (seamIndex < numSeams) {
        seamsStart[seamIndex] = lastMinEnergySums[minSumIndex].xPos;
        seamIndex++;
        minSumIndex = (minSumIndex + 1) % width;
    }

    //sort by coordinate
    quicksortint(seamsStart, 0, numSeams - 1);
    cudaStatus = hipMemcpy(d_seams + numSeams * (height - 1), seamsStart, seamStartSize_t, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "Memory Copy seamsStart -> d_seams failed! ErrorCode %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus)); }

    //start kernel3 calcSeams
    dim3 threadsPerBlock3(blocksize3);
    dim3 numBlocks3(ceil(numSeams/ (double)blocksize3));
    printf("\n%d %d\n", threadsPerBlock3.x, numBlocks3.x);
    calcSeams<<<numBlocks3, threadsPerBlock3 >>>(d_minEnergySums, d_seams, width, height, numSeams);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) { fprintf(stderr, "calcSeams launch failed: %s\n", hipGetErrorString(cudaStatus)); }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize after launch calcSeams failed: %s\n", hipGetErrorString(cudaStatus)); }

    //start kernel4 increaseWidth
    dim3 threadsPerBlock4(blocksize4);
    dim3 numBlocks4(ceil(height / (double)blocksize4));
    increaseWidth<<<numBlocks4, threadsPerBlock4>>>(d_inputData, d_outputData, d_minEnergySums, d_seams, width, height, numSeams);
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) { fprintf(stderr, "increaseWidth launch failed: %s\n", hipGetErrorString(cudaStatus)); }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize after launch increaseWidth failed: %s\n", hipGetErrorString(cudaStatus)); }

    //copy outputData and create image
    cudaStatus = hipMemcpy(outputData, d_outputData, outputDataSize_t, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) { fprintf(stderr, "Memory Copy d_outputData -> outputData failed! ErrorCode %d: %s\n", cudaStatus, hipGetErrorString(cudaStatus)); }
    input->width = width + numSeams;
    input->lpData = outputData;

    //free Memory
    hipFree(&d_inputData);
    hipFree(&d_outputData);
    hipFree(&d_pixelEnergies);
    hipFree(&d_minEnergySums);
    hipFree(&d_seams);

    clock_t end = clock();
    printf("Execution time: %4.2f sec\n", (double)((double)(end - start) / CLOCKS_PER_SEC));
    storeJpegImageFile(input, outputFile);
    free(seamsStart);
    free(lastMinEnergySums);
    free(outputData);
    return 0;
}
